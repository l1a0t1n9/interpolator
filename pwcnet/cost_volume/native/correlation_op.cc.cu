
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA

#define EIGEN_USE_GPU

#include "tensorflow/core/framework/tensor_types.h"
#include "tensorflow/core/platform/types.h"
#include "tensorflow/core/util/cuda_kernel_helper.h"

#include "correlation_op.h"

using namespace tensorflow;
using CPUDevice = Eigen::ThreadPoolDevice;
using GPUDevice = Eigen::GpuDevice;

// ---------------------------------------------------------
// DIRECT PORT OF CAFFE CODE WITH MINIMAL CHANGES
// ---------------------------------------------------------
  
#define ROUND_OFF 50000

#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

const int CAFFE_CUDA_NUM_THREADS = 512;

inline int CAFFE_GET_BLOCKS(const int N) {
  return (N + CAFFE_CUDA_NUM_THREADS - 1) / CAFFE_CUDA_NUM_THREADS;
}

// Adds padding by moving the elements in the array around.
template <typename Dtype>
__global__ void RearrangeWithPadding(const Dtype* in, Dtype* out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
  int xy = blockIdx.y * gridDim.x + blockIdx.x;
  int ch = threadIdx.x;
  int n  = blockIdx.z;

  int index = (n * widthheight + xy) * channels + ch;
  Dtype value = in[index];

  __syncthreads();

  // Compute the new position of this pixel, after padding has been added.
  int xpad  = (xy % width + padding);
  int ypad  = (xy / width + padding);
  int xypad = ypad * (width+2*padding) + xpad;

  out[(n*pwidthheight+xypad)*channels + ch] = value;
}


// High level outline:
// Let c0 be the features over which correlation neighborhoods are centered on, and c1 be the other features.
// Each block is responsible for computing the correlations across the input channel dimension.
// 1. Load the kernel-size dependent patches in c0 into shared memory for the block.
// 2. Each thread (they are spread across the input channel dimension) in the block iterates 
//    over the spatial neighborhood in c1 to compute correlations.
// 3. Finally, we use 1 thread in the block to reduce sum over the input channel dimension,
//    by gathering summed results from the other threads.
//
// Note that experimentally WARPS_PER_BLOCK should be 1 for optimal performance. 
// The reduction in performance for larger values might be due to increased memory pressure as the 
// kernel is bottlenecked by reads from global memory.
template <typename Dtype>
__global__ void CorrelateData(const int nthreads, int num, int topwidth, int topheight, int topchannels, int topcount,
  int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int kernel_size, int stride1, int stride2,
  int bottomwidth, int bottomheight, int bottomchannels,
  const Dtype *bottom0, const Dtype *bottom1, Dtype *top)
{
  extern __shared__ char patch_data_char[];

  Dtype *patch_data = (Dtype *)patch_data_char;

  // Kernel for c0 is not centered. Its top left is always at the center regardless of kernel size.
  int x1 = blockIdx.x*stride1 + max_displacement;
  int y1 = blockIdx.y*stride1 + max_displacement;
  int item = blockIdx.z;
  int ch_off = threadIdx.x;

  // Load 3D patch into shared memory.
  // Threads are spread across the channel dimension (i.e thread0, ... threadN, thread0, ... threadN, ...).
  for(int j = 0; j < kernel_size; j++) { // HEIGHT
    for(int i = 0; i < kernel_size; i++) { // WIDTH
      int ji_off = ((j * kernel_size) + i) * bottomchannels;
      for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
        int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + ch;
        int idxPatchData = ji_off + ch;
        patch_data[idxPatchData] = bottom0[idx1];
      }
    }
  }

  __syncthreads();

  __shared__ Dtype sum[WARPS_PER_BLOCK*THREADS_PER_WARP];

  // For this input channel, compute correlation over the spatial neighborhood. 
  // top_channel (output channel) is an index into the spatial neighborhood.
  for(int top_channel = 0; top_channel < topchannels; top_channel++) {
    sum[ch_off] = 0;

    // Compute x and y offsets from neighborhood center.
    int s2o = (top_channel % neighborhood_grid_width - neighborhood_grid_radius) * stride2;
    int s2p = (top_channel / neighborhood_grid_width - neighborhood_grid_radius) * stride2;

    for(int j = 0; j < kernel_size; j++) { // HEIGHT
      for(int i = 0; i < kernel_size; i++) { // WIDTH
        int ji_off = ((j * kernel_size) + i) * bottomchannels;
        for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
          int x2 = x1 + s2o;
          int y2 = y1 + s2p;

          int idxPatchData = ji_off + ch;
          int idx2 = ((item * bottomheight + y2+j) * bottomwidth + x2+i) * bottomchannels + ch;
          
          sum[ch_off] += patch_data[idxPatchData] * bottom1[idx2];
        }
      }
    }

    // Reduce sum over the input channel dimension, using the thread at channel idx 0.
    __syncthreads();

    if(ch_off == 0) {
        Dtype total_sum = 0;
#pragma unroll
        for(int idx = 0; idx < WARPS_PER_BLOCK*THREADS_PER_WARP; idx++) {
          total_sum += sum[idx];
        }

        const int sumelems = kernel_size*kernel_size*bottomchannels;
        const int index = (blockIdx.y * topwidth + blockIdx.x) * topchannels + top_channel;
        top[index + item*topcount] = total_sum / (float)sumelems;
    }
  }
}

template <typename Dtype, int kernel_size>
__global__ void CorrelateDataUnrolled(const int nthreads, int num, int topwidth, int topheight, int topchannels, int topcount,
	int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
	int bottomwidth, int bottomheight, int bottomchannels,
	const Dtype *bottom0, const Dtype *bottom1, Dtype *top)
{
	extern __shared__ char patch_data_char[];

	Dtype *patch_data = (Dtype *)patch_data_char;

	// Kernel for c0 is not centered. Its top left is always at the center regardless of kernel size.
	int x1 = blockIdx.x*stride1 + max_displacement;
	int y1 = blockIdx.y*stride1 + max_displacement;
	int item = blockIdx.z;
	int ch_off = threadIdx.x;

	// Load 3D patch into shared memory.
	// Threads are spread across the channel dimension (i.e thread0, ... threadN, thread0, ... threadN, ...).
#pragma unroll
	for (int j = 0; j < kernel_size; j++) { // HEIGHT
#pragma unroll
		for (int i = 0; i < kernel_size; i++) { // WIDTH
			int ji_off = ((j * kernel_size) + i) * bottomchannels;
			for (int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
				int idx1 = ((item * bottomheight + y1 + j) * bottomwidth + x1 + i) * bottomchannels + ch;
				int idxPatchData = ji_off + ch;
				patch_data[idxPatchData] = bottom0[idx1];
			}
		}
	}

	__syncthreads();

	__shared__ Dtype sum[WARPS_PER_BLOCK*THREADS_PER_WARP];

	// For this input channel, compute correlation over the spatial neighborhood. 
	// top_channel (output channel) is an index into the spatial neighborhood.
	for (int top_channel = 0; top_channel < topchannels; top_channel++) {
		sum[ch_off] = 0;

		// Compute x and y offsets from neighborhood center.
		int s2o = (top_channel % neighborhood_grid_width - neighborhood_grid_radius) * stride2;
		int s2p = (top_channel / neighborhood_grid_width - neighborhood_grid_radius) * stride2;

#pragma unroll
		for (int j = 0; j < kernel_size; j++) { // HEIGHT
#pragma unroll
			for (int i = 0; i < kernel_size; i++) { // WIDTH
				int ji_off = ((j * kernel_size) + i) * bottomchannels;
				for (int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK*THREADS_PER_WARP)) { // CHANNELS
					int x2 = x1 + s2o;
					int y2 = y1 + s2p;

					int idxPatchData = ji_off + ch;
					int idx2 = ((item * bottomheight + y2 + j) * bottomwidth + x2 + i) * bottomchannels + ch;

					sum[ch_off] += patch_data[idxPatchData] * bottom1[idx2];
				}
			}
		}

		// Reduce sum over the input channel dimension, using the thread at channel idx 0.
		__syncthreads();

		if (ch_off == 0) {
			Dtype total_sum = 0;
#pragma unroll
			for (int idx = 0; idx < WARPS_PER_BLOCK*THREADS_PER_WARP; idx++) {
				total_sum += sum[idx];
			}

			const int sumelems = kernel_size*kernel_size*bottomchannels;
			const int index = (blockIdx.y * topwidth + blockIdx.x) * topchannels + top_channel;
			top[index + item*topcount] = total_sum / (float)sumelems;
		}
	}
}

// Notes on gradient computation kernels:
// - bottom holds gradients dl/dx that we need to compute. topdiff holds gradients dl/dy.
// - xmin, xmax, ymin, ymax index into topdiff (i.e they define the range for which the output is affected by dx).
// - For a kernel size larger than 1, we need to shift around topdiff, although the underlying dy/dx is the same.

// Computes the gradients with respect to c0.
template <typename Dtype>
__global__ void CorrelateDataBackward0(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  Dtype *bottom0diff, const Dtype *bottom1, const Dtype *topdiff)
{
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index % bottomchannels; //channels
    int l = (index / bottomchannels) % bottomwidth + pad_size; //w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight + pad_size; //h-pos

    // Get X,Y ranges and clamp
    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;

    // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
    int xmin = (l - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1
    int ymin = (m - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement) / stride1

    // Same here:
    int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off; // floor (l - max_displacement) / stride1
    int ymax = (m - max_displacement + round_off_s1) / stride1 - round_off; // floor (m - max_displacement) / stride1


    Dtype sum = 0;
    if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
    {
        xmin = max(0,xmin);
        xmax = min(topwidth-1,xmax);

        ymin = max(0,ymin);
        ymax = min(topheight-1,ymax);

        for(int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
          for(int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {

            // Get bottom1 data:
            int s2o = stride2 * o;
            int s2p = stride2 * p;
            int idxbot1 = ((item * pbottomheight + (m+s2p)) * pbottomwidth + (l+s2o)) * bottomchannels + n;
            Dtype bot1tmp = bottom1[idxbot1]; // bottom1[l+s2o,m+s2p,n]

            // Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * neighborhood_grid_width + (o+neighborhood_grid_radius); // index [o,p]

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = ((item * topheight + y) * topwidth + x) * topchannels + op;
                sum += topdiff[idxtopdiff] * bot1tmp;
              }
            }
          }
        }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    const int bot0index = ((m - pad_size) * bottomwidth + (l - pad_size)) * bottomchannels + n;
    bottom0diff[bot0index + item*bottomcount] = sum / (float)sumelems;
  }

}

// Computes the gradients with respect to c1.
// Note that the structure of the code is almost identical to the gradient for c0, 
// except that our indexing into topdiff moves around a lot more (in addition to the shifting for the kernel).
template <typename Dtype>
__global__ void CorrelateDataBackward1(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels,
  int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2,
  int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size,
  const Dtype *bottom0, Dtype *bottom1diff, const Dtype *topdiff)
{
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index % bottomchannels; //channels
    int l = (index / bottomchannels) % bottomwidth + pad_size; //w-pos
    int m = (index / bottomchannels / bottomwidth) % bottomheight + pad_size; //h-pos

    // round_off is a trick to enable integer division with ceil, even for negative numbers
    // We use a large offset, for the inner part not to become negative.
    const int round_off = ROUND_OFF;
    const int round_off_s1 = stride1 * round_off;

    Dtype sum = 0;
    for(int p = -neighborhood_grid_radius; p <= neighborhood_grid_radius; p++) {
      for(int o = -neighborhood_grid_radius; o <= neighborhood_grid_radius; o++) {

        int s2o = stride2 * o;
        int s2p = stride2 * p;

        // Get X,Y ranges and clamp
        // We add round_off before_s1 the int division and subtract round_off after it, to ensure the formula matches ceil behavior:
        int xmin = (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1
        int ymin = (m - 2*kernel_radius - max_displacement - s2p + round_off_s1 - 1) / stride1 + 1 - round_off; // ceil (l - 2*kernel_radius - max_displacement - s2o) / stride1

        // Same here:
        int xmax = (l - max_displacement - s2o + round_off_s1) / stride1 - round_off; // floor (l - max_displacement - s2o) / stride1
        int ymax = (m - max_displacement - s2p + round_off_s1) / stride1 - round_off; // floor (m - max_displacement - s2p) / stride1

        if(xmax>=0 && ymax>=0 && (xmin<=topwidth-1) && (ymin<=topheight-1))
        {
            xmin = max(0,xmin);
            xmax = min(topwidth-1,xmax);

            ymin = max(0,ymin);
            ymax = min(topheight-1,ymax);

            // Get bottom0 data:
            int idxbot0 = ((item * pbottomheight + (m-s2p)) * pbottomwidth + (l-s2o)) * bottomchannels + n;
            Dtype bot0tmp = bottom0[idxbot0]; // bottom1[l+s2o,m+s2p,n]

            // Index offset for topdiff in following loops:
            int op = (p+neighborhood_grid_radius) * neighborhood_grid_width + (o+neighborhood_grid_radius); // index [o,p]

            for(int y = ymin; y <= ymax; y++) {
              for(int x = xmin; x <= xmax; x++) {
                int idxtopdiff = ((item * topheight + y) * topwidth + x) * topchannels + op;
                sum += topdiff[idxtopdiff] * bot0tmp;
              }
            }
        }
      }
    }
    const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
    const int bot1index = ((m - pad_size) * bottomwidth + (l - pad_size)) * bottomchannels + n;
		bottom1diff[bot1index + item*bottomcount] = sum / (float)sumelems;
  }
}

void Correlation(const GPUDevice& d,
                 typename TTypes<float, 4>::ConstTensor input_0,
                 typename TTypes<float, 4>::ConstTensor input_1,
                 typename TTypes<float, 4>::Tensor output,
                 typename TTypes<float, 4>::Tensor padded_0,
                 typename TTypes<float, 4>::Tensor padded_1,
                 CorrelationState st) {

  const int top_channels_ = output.dimension(3);
  const int top_height_ = output.dimension(1);
  const int top_width_ = output.dimension(2);
  const int pad_size_ = st.pad_size;
  const int stride1_ = st.stride_1;
  const int stride2_ = st.stride_2;
  const int kernel_size_ = st.kernel_size;
  const int kernel_radius_ = st.kernel_radius;
  const int max_displacement_ = st.max_displacement;
  const int neighborhood_grid_radius_ = st.neighborhood_grid_radius;
  const int neighborhood_grid_width_ = st.neighborhood_grid_width;

  // PORTED CAFFE CODE

  const int bnum = input_0.dimension(0);
  const int bchannels = input_0.dimension(3);
  const int bheight = input_0.dimension(1);
  const int bwidth = input_0.dimension(2);
  const int bwidthheight = bwidth * bheight;

  const int topcount = top_width_ * top_height_ * top_channels_;

  dim3 threadsPerBlock(THREADS_PER_WARP * WARPS_PER_BLOCK);

  cudaMemset(padded_0.data(), 0, padded_0.size()*sizeof(float));
  cudaMemset(padded_1.data(), 0, padded_1.size()*sizeof(float));

  int threads_per_block = bchannels;
  dim3 totalBlocksRearr(bwidth, bheight, bnum);
  const int pwidthheight = (bwidth + 2 * pad_size_) * (bheight + 2 * pad_size_);

  RearrangeWithPadding<float><<<totalBlocksRearr,threads_per_block>>>
          (input_0.data(),padded_0.data(),bnum,bchannels,bwidth,bheight,bwidthheight,pad_size_,pwidthheight);

  RearrangeWithPadding<float><<<totalBlocksRearr,threads_per_block>>>
          (input_1.data(),padded_1.data(),bnum,bchannels,bwidth,bheight,bwidthheight,pad_size_,pwidthheight);

  const int num = bnum;
  const int channels = bchannels;
  const int height = bheight + 2*pad_size_;
  const int width = bwidth + 2*pad_size_;

  const int shared_memory_per_block = (kernel_size_*kernel_size_)*bchannels;

  // CorrelationLayer
  int topThreadCount = topcount;

  dim3 totalBlocksCorr(top_width_, top_height_, num);

  if (kernel_size_ == 1) {
	  CorrelateDataUnrolled<float, 1> << <totalBlocksCorr, threadsPerBlock, shared_memory_per_block * sizeof(float) >> >(
		  topThreadCount,
		  num, top_width_, top_height_, top_channels_, topcount,
		  max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
		  stride1_, stride2_,
		  width, height, channels,
		  padded_0.data(), padded_1.data(), output.data()
		  );
  }
  else if (kernel_size_ == 2) {
	  CorrelateDataUnrolled<float, 2> << <totalBlocksCorr, threadsPerBlock, shared_memory_per_block * sizeof(float) >> >(
		  topThreadCount,
		  num, top_width_, top_height_, top_channels_, topcount,
		  max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
		  stride1_, stride2_,
		  width, height, channels,
		  padded_0.data(), padded_1.data(), output.data()
		  );
  }
  else if (kernel_size_ == 3) {
	  CorrelateDataUnrolled<float, 3> << <totalBlocksCorr, threadsPerBlock, shared_memory_per_block * sizeof(float) >> >(
		  topThreadCount,
		  num, top_width_, top_height_, top_channels_, topcount,
		  max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
		  stride1_, stride2_,
		  width, height, channels,
		  padded_0.data(), padded_1.data(), output.data()
		  );
  }
  else {
	  CorrelateData<float> << <totalBlocksCorr, threadsPerBlock, shared_memory_per_block * sizeof(float) >> >(
		  topThreadCount,
		  num, top_width_, top_height_, top_channels_, topcount,
		  max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_, kernel_size_,
		  stride1_, stride2_,
		  width, height, channels,
		  padded_0.data(), padded_1.data(), output.data()
		  );
  }
}

void CorrelationGrad(const GPUDevice& d,
                     typename TTypes<float, 4>::ConstTensor input_grad,
                     typename TTypes<float, 4>::ConstTensor padded_0,
                     typename TTypes<float, 4>::ConstTensor padded_1,
                     typename TTypes<float, 4>::Tensor output_grad_0,
                     typename TTypes<float, 4>::Tensor output_grad_1,
                     CorrelationState st) {

  const int top_channels_ = input_grad.dimension(3);
  const int top_height_ = input_grad.dimension(1);
  const int top_width_ = input_grad.dimension(2);

  const int pad_size_ = st.pad_size;
  const int stride1_ = st.stride_1;
  const int stride2_ = st.stride_2;
  const int kernel_radius_ = st.kernel_radius;
  const int max_displacement_ = st.max_displacement;
  const int neighborhood_grid_radius_ = st.neighborhood_grid_radius;
  const int neighborhood_grid_width_ = st.neighborhood_grid_width;

  // PORTED CAFFE CODE

  // Get top diff, compute bottom diff
  const float* top_diff = input_grad.data();

  float* bottom0_diff = output_grad_0.data();
  float* bottom1_diff = output_grad_1.data();

  const int num = output_grad_0.dimension(0);
  const int channels = output_grad_0.dimension(3);
  const int height = output_grad_0.dimension(1);
  const int width = output_grad_0.dimension(2);

  const int paddedheight = height + 2*pad_size_;
  const int paddedwidth = width + 2*pad_size_;

  const int bottomcount = channels * height * width;

  int botThreadCount = bottomcount;

  // CorrelationLayerBackward

  // == Run kernel Backward 0
  dim3 totalBlocksBackward0(width, height, channels * num); //First dim is fastest
  dim3 threadsPerBlockBackward0(THREADS_PER_WARP * WARPS_PER_BLOCK);
  const int buffer_size_backw0 = ((int)ceil((float)(2 * kernel_radius_) / (float)stride1_) + 1) * top_channels_;

  // == Run kernel Backward 0
  for(int n = 0; n < num; n++) {
  //Bottom0:
  CorrelateDataBackward0<float><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
      botThreadCount,
      num, n, top_width_, top_height_, top_channels_,
      max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
      stride1_, stride2_,
      width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
      bottom0_diff, padded_1.data(), top_diff
      );
  }

  // == Run kernel Backward 1
  for(int n = 0; n < num; n++) {
  CorrelateDataBackward1<float><<<CAFFE_GET_BLOCKS(botThreadCount), CAFFE_CUDA_NUM_THREADS>>>(
      botThreadCount,
      num, n, top_width_, top_height_, top_channels_,
      max_displacement_, neighborhood_grid_radius_, neighborhood_grid_width_, kernel_radius_,
      stride1_, stride2_,
      width, height, paddedwidth, paddedheight, channels, bottomcount, pad_size_,
      padded_0.data(), bottom1_diff, top_diff
      );
  }

}

#endif  // GOOGLE_CUDA